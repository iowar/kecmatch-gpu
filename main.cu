#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>

#include "common.cuh"
#include "sha3.cuh"

#define BLOCKS 16
#define THREADS 256

#define N 1000000000

__device__ uint8_t dev_wanted_signature[4] = {0x0, 0x0, 0x0, 0x0};

__global__ void init_signature(uint32_t *fn_sig) {
    dev_wanted_signature[0] = *fn_sig >> 24;
    dev_wanted_signature[1] = ((*fn_sig >> 16) & 0xff);
    dev_wanted_signature[2] = ((*fn_sig >> 8) & 0xff);
    dev_wanted_signature[3] = ((*fn_sig >> 0) & 0xff);
}

__global__ void calculate(char *fn_name, char *fn_args) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init((unsigned long long)clock() + tid, 0, 0, &state);
    int block = (int)(hiprand_uniform_double(&state) * 1000000);

    int index = 0;
    char id[16];
    char data[128];

    memset(data, 0, 128);
    memcpy(data, fn_name, _strlen(fn_name));
    _itoa(block, id, 10);
    memcpy(&data[_strlen(data)], id, _strlen(id));
    memcpy(&data[_strlen(data)], "000000000", 9);
    index = _strlen(data);
    memcpy(&data[_strlen(data)], fn_args, _strlen(fn_args));

    for (int i = 0; i < N; i++) {
	_itoa(i, id, 10);
	memcpy(&data[index - _strlen(id)], id, _strlen(id));

	uint8_t hash[64];
	sha3_return_t ok = sha3_HashBuffer(256, SHA3_FLAGS_KECCAK, data,
					   _strlen(data), hash, 64);
	if (ok != 0) {
	    printf("bad params\n");
	    return;
	}

	if (hash[0] == dev_wanted_signature[0] &&
	    hash[1] == dev_wanted_signature[1] &&
	    hash[2] == dev_wanted_signature[2] &&
	    hash[3] == dev_wanted_signature[3]) {
	    printf(
		"thread => %d method => %s method id => 0x%02x%02x%02x%02x\n",
		tid, data, hash[0], hash[1], hash[2], hash[3]);
	}
    }
}

// TODO add speed
void *metrics(void *data) {
    char spin[4] = {'-', '\\', '|', '/'};

    while (1) {
	for (int i = 0; i < 1000000; i++) {
	    usleep(100000);
	    printf("\33[2K\r Searching %c ", spin[i % 4]);
	}
    }
    return NULL;
}

int main(int argc, char **argv) {
    int opt;
    char *fvalue = NULL;  // function name
    char *avalue = NULL;  // arguments
    char *svalue = NULL;  // signature
    uint32_t signature;

    while ((opt = getopt(argc, argv, "f:a:s:")) != -1) {
	switch (opt) {
	    case 'f':
		fvalue = optarg;
		break;
	    case 'a':
		avalue = optarg;
		break;
	    case 's':
		svalue = optarg;
		if (svalue[0] != '0' || svalue[1] != 'x' ||
		    _strlen(svalue) != 10) {
		    fprintf(stderr, "Wrong signature format!\n");
		    exit(EXIT_FAILURE);
		}

		signature = strtoul(svalue + 2, NULL, 16);
		break;
	    default: /* '?' */
		usage(argv[0]);
		exit(EXIT_FAILURE);
	}
    }

    if (fvalue == NULL || avalue == NULL || svalue == NULL) {
	usage(argv[0]);
	exit(EXIT_FAILURE);
    }

    pthread_t th;
    int ret = pthread_create(&th, NULL, &metrics, NULL);
    if (ret != 0) {
	printf("Error: pthread_create() failed\n");
	return 1;
    }

    // device arguments
    char *dev_f, *dev_a;
    uint32_t *dev_s;

    HANDLE_ERROR(hipMalloc((void **)&dev_f, _strlen(fvalue) * sizeof(char)));
    HANDLE_ERROR(hipMalloc((void **)&dev_a, _strlen(avalue) * sizeof(char)));
    HANDLE_ERROR(hipMalloc((void **)&dev_s, sizeof(uint32_t)));

    HANDLE_ERROR(hipMemcpy(dev_f, fvalue, _strlen(fvalue) * sizeof(char),
			    hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_a, avalue, _strlen(avalue) * sizeof(char),
			    hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_s, &signature, sizeof(uint32_t),
			    hipMemcpyHostToDevice));

    init_signature<<<1, 1>>>(dev_s);
    HANDLE_ERROR(hipFree(dev_s));

    calculate<<<BLOCKS, THREADS>>>(dev_f, dev_a);
    hipDeviceSynchronize();  // not important

    HANDLE_ERROR(hipFree(dev_f));
    HANDLE_ERROR(hipFree(dev_a));
    exit(EXIT_SUCCESS);
}
